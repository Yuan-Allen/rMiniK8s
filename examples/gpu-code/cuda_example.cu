#include <chrono>
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>

using namespace std;

#define BLOCK_NUM 4
#define THREAD_NUM 4
#define R_SIZE BLOCK_NUM *THREAD_NUM
#define M_SIZE R_SIZE *R_SIZE

#define CHECK(call)                                                            \
    {                                                                          \
        const hipError_t error = call;                                        \
        if (error != hipSuccess) {                                            \
            printf("ERROR: %s:%d,", __FILE__, __LINE__);                       \
            printf("code:%d,reason:%s\n", error, hipGetErrorString(error));   \
            exit(1);                                                           \
        }                                                                      \
    }

void initDevice(int devNum) {
    int dev = devNum;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("Using device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));
}

void initialData(float *ip, int size) {
    time_t t;
    srand((unsigned)time(&t));
    for (int i = 0; i < size; i++) {
        ip[i] = (float)(rand() & 0xffff) / 1000.0f;
    }
}

void checkResult(float *hostRef, float *gpuRef, const int N) {
    double epsilon = 1.0E-8;
    for (int i = 0; i < N; i++) {
        if (abs(hostRef[i] - gpuRef[i]) > epsilon) {
            printf("Results don\'t match!\n");
            printf("%f(hostRef[%d] )!= %f(gpuRef[%d])\n", hostRef[i], i,
                   gpuRef[i], i);
            return;
        }
    }
    printf("Check result success!\n");
}

void sumMatrix2DonCPU(float *MatA, float *MatB, float *MatC, int nx, int ny) {
    float *a = MatA;
    float *b = MatB;
    float *c = MatC;
    for (int j = 0; j < ny; j++) {
        for (int i = 0; i < nx; i++) {
            c[i] = a[i] + b[i];
        }
        c += nx;
        b += nx;
        a += nx;
    }
}

__global__ void sumMatrix(float *MatA, float *MatB, float *MatC, int nx,
                          int ny) {
    int ix = threadIdx.x + blockDim.x * blockIdx.x;
    int iy = threadIdx.y + blockDim.y * blockIdx.y;
    int idx = ix + iy * nx;
    if (ix < nx && iy < ny) {
        MatC[idx] = MatA[idx] + MatB[idx];
    }
}

__global__ void mat_mul(int *mat1, int *mat2, int *result) {
    const int bid = blockIdx.x;
    const int tid = threadIdx.x;
    const int row = bid * THREAD_NUM + tid;
    for (int c = 0; c < R_SIZE; c++) {
        for (int n = 0; n < R_SIZE; n++) {
            result[row * R_SIZE + c] +=
                mat1[row * R_SIZE + n] * mat2[n * R_SIZE + c];
        }
    }
}

void testAdd() {
    printf("test add...\n");
    initDevice(0);

    int nx = 1 << 12;
    int ny = 1 << 12;
    int nBytes = nx * ny * sizeof(float);

    float *A_host = (float *)malloc(nBytes);
    float *B_host = (float *)malloc(nBytes);
    float *C_host = (float *)malloc(nBytes);
    float *C_from_gpu = (float *)malloc(nBytes);
    initialData(A_host, nx * ny);
    initialData(B_host, nx * ny);

    float *A_dev = NULL;
    float *B_dev = NULL;
    float *C_dev = NULL;
    CHECK(hipMalloc((void **)&A_dev, nBytes));
    CHECK(hipMalloc((void **)&B_dev, nBytes));
    CHECK(hipMalloc((void **)&C_dev, nBytes));

    CHECK(hipMemcpy(A_dev, A_host, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(B_dev, B_host, nBytes, hipMemcpyHostToDevice));

    dim3 threadsPerBlock(32, 32);

    dim3 numBlocks((nx - 1) / threadsPerBlock.x + 1,
                   (ny - 1) / threadsPerBlock.y + 1);

    auto beforeTime = std::chrono::steady_clock::now();
    sumMatrix<<<numBlocks, threadsPerBlock>>>(A_dev, B_dev, C_dev, nx, ny);
    auto afterTime = std::chrono::steady_clock::now();
    double duration_millsecond =
        std::chrono::duration<double, std::milli>(afterTime - beforeTime)
            .count();
    CHECK(hipDeviceSynchronize());

    printf("GPU Execution Time: %f ms\n", duration_millsecond);

    CHECK(hipMemcpy(C_from_gpu, C_dev, nBytes, hipMemcpyDeviceToHost));
    beforeTime = std::chrono::steady_clock::now();
    sumMatrix2DonCPU(A_host, B_host, C_host, nx, ny);
    afterTime = std::chrono::steady_clock::now();
    duration_millsecond =
        std::chrono::duration<double, std::milli>(afterTime - beforeTime)
            .count();

    printf("CPU Execution Time: %f ms\n", duration_millsecond);

    checkResult(C_host, C_from_gpu, nx * ny);

    hipFree(A_dev);
    hipFree(B_dev);
    hipFree(C_dev);
    free(A_host);
    free(B_host);
    free(C_host);
    free(C_from_gpu);
    hipDeviceReset();
}

void testMul() {
    printf("test mul...\n");
    initDevice(0);

    int *mat1, *mat2, *result;
    int *g_mat1, *g_mat2, *g_mat_result;

    // 1-dim NxN vector to represent 2-dim (N, N) matrix
    mat1 = (int *)malloc(M_SIZE * sizeof(int));
    mat2 = (int *)malloc(M_SIZE * sizeof(int));
    result = (int *)malloc(M_SIZE * sizeof(int));
    printf("M_SIZE:%d\n", M_SIZE);
    // init matrices
    for (int i = 0; i < M_SIZE; i++) {
        mat1[i] = rand() % 10;
        mat2[i] = rand() % 10;
        result[i] = 0;
    }
    hipMalloc((void **)&g_mat1, sizeof(int) * M_SIZE);
    hipMalloc((void **)&g_mat2, sizeof(int) * M_SIZE);
    hipMalloc((void **)&g_mat_result, sizeof(int) * M_SIZE);
    hipMemcpy(g_mat1, mat1, sizeof(int) * M_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(g_mat2, mat2, sizeof(int) * M_SIZE, hipMemcpyHostToDevice);
    mat_mul<<<BLOCK_NUM, THREAD_NUM>>>(g_mat1, g_mat2, g_mat_result);
    hipMemcpy(result, g_mat_result, sizeof(int) * M_SIZE,
               hipMemcpyDeviceToHost);
    printf("res[0]:%d\n", result[0]);
}

int main(int argc, char **argv) {
    testAdd();
    testMul();
    return 0;
}